#include "hip/hip_runtime.h"
#include "lab3_cuda.h"
#include <math.h>
#include <stdbool.h>
#include <assert.h>
#include <string.h>
#include <time.h>
#include <chrono>
using namespace std::chrono;

#define EPSILON 1e-4
#define THRESHOLD 1e-4
#define MAX_BLOCK_SIZE 1024
#define MAX_SWEEPS 30
#define MAX_ITER 10000000
#define MULTIPLY_BLOCK_SIZE 64

#pragma GCC optimize("Ofast")
#pragma GCC target("sse,sse2,sse3,ssse3,sse4,popcnt,abm,mmx,avx,tune=native")

#define gpuErrchk(ans)                        \
    {                                         \
        gpuAssert((ans), __FILE__, __LINE__); \
    }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort = true)
{
    if (code != hipSuccess)
    {
        fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
        if (abort)
            exit(code);
    }
}

void s_initialize_identity(double *I, int size)
{
    memset(I, 0, sizeof(double)*size*size);
    for (int i = 0; i < size; i++)
        I[i * size + i] = 1.0;
}

void s_transpose(double *M, int m, int n, double *M_T)
{
    int i, j, index_;
    for (j=0; j<n; j++)
    {
        index_ = j*m;
        for (i=0; i<m; i++)
        {
            M_T[index_+i] = M[i*n+j];
        }
    }
}

void s_multiply(double *M_1, int m1, int n1, double *M_2, int m2, int n2, double *result)
{
    assert(n1 == m2);
    double sum = 0.0;
    //compute M_2_T:
    double *M_2_T = (double *)malloc(sizeof(double) * n2 * m2);
    s_transpose(M_2, m2, n2, M_2_T);
    int i, j, k, temp1, temp2;
    for (i = 0; i < m1; i++)
    {
        temp1 = i * n1;
        for (j = 0; j < n2; j++)
        {
            sum = 0.0;
            temp2 = j * m2;
            for (k = 0; k < n1; k++)
            {
                sum += M_1[temp1 + k] * M_2_T[temp2 + k];
            }
            result[i * n2 + j] = sum;
        }
    }
    free(M_2_T);
}

double s_maxind(double *A, int size, int k)
{
    int m = k + 1;
    for (int i = k + 2; i < size; i++)
    {
        if (fabs(A[k * size + i]) > fabs(A[k * size + m]))
        {
            m = i;
        }
    }
    return m;
}

void s_update(int k, double t, double *e, bool *changed, int *state)
{
    double y = e[k];
    e[k] = y + t;
    if (changed[k] && (y == e[k]))
    {
        changed[k] = false;
        (*state)--;
    }
    else if (!changed[k] && (y != e[k]))
    {
        changed[k] = true;
        (*state)++;
    }
}

void s_rotate(int k, int l, int i, int j, double *A, int P, double c, double s)
{
    double k_l = c * A[k * P + l] - s * A[i * P + j];
    double i_j = s * A[k * P + l] + c * A[i * P + j];
    A[k * P + l] = k_l;
    A[i * P + j] = i_j;
}

void s_merge(double *e, int *indices_e, int left_index, int mid, int right_index)
{
    int i = left_index, j = mid + 1, k = 0;
    double *sorted = (double *)malloc(sizeof(double) * (right_index - left_index + 1));
    int *sorted_indices = (int *)malloc(sizeof(int) * (right_index - left_index + 1));
    assert(sorted_indices!=NULL);
    while (i <= mid && j <= right_index)
    {
        if (fabs(e[i]) >= fabs(e[j]))
        {
            sorted_indices[k] = indices_e[i];
            sorted[k++] = e[i++];
        }
        else
        {
            sorted_indices[k] = indices_e[j];
            sorted[k++] = e[j++];
        }
    }
    while (i <= mid)
    {
        sorted_indices[k] = indices_e[i];
        sorted[k++] = e[i++];
    }
    while (j <= right_index)
    {
        sorted_indices[k] = indices_e[j];
        sorted[k++] = e[j++];
    }
    assert(k==right_index-left_index+1);
    memcpy(e + left_index, sorted, sizeof(double)*(right_index-left_index+1));
    memcpy(indices_e + left_index, sorted_indices, sizeof(int)*(right_index-left_index+1));
    free(sorted);
    free(sorted_indices);
}

void s_mergesort(double *e, int e_len, int *indices_e, int left_index, int right_index)
{
    //sort e in desc based on abs value
    //rearrange corresponding indices_e appropriately
    assert(left_index <= right_index);
    if (left_index < right_index)
    {
        int mid = (left_index + right_index) / 2;
        s_mergesort(e, e_len, indices_e, left_index, mid);
        s_mergesort(e, e_len, indices_e, mid + 1, right_index);
        s_merge(e, indices_e, left_index, mid, right_index);
    }
}

double l2_matrix_diff_norm(double *E_, double *E, int M, int N)
{
    double sum = 0.0;
    for (int i = 0; i < M; i++)
    {
        for (int j = 0; j < N; j++)
            sum += (E_[i * M + j] - E[i * M + j]) * (E_[i * M + j] - E[i * M + j]);
    }
    return sqrt(sum);
}

double l2_diff_norm(double *e_, double *e, int len)
{
    double sum = 0.0;
    for (int i = 0; i < len; i++)
    {
        sum += (e_[i] - e[i]) * (e_[i] - e[i]);
    }
    return sqrt(sum);
}

void print_matrix(double *A, int M, int N, bool console)
{
    return;
    for (int i = 0; i < M; i++)
    {
        for (int j = 0; j < N; j++)
        {
            if (!console)
                fprintf(stderr, "%f ", A[i * N + j]);
            else
                printf("%f ", A[i * N + j]);
        }
        if (!console)
            fprintf(stderr, "\n");
        else
            printf("\n");
    }
}

void s_compute_V(double **SIGMA, double *D_T, double **U, double **V_T, int N, int P)
{
    //V_T = INV-SIGMA * U_T * M
    double *INV_SIGMA = (double *)calloc(N * P, sizeof(double)); //|=NXP
    for (int i = 0; i < P; i++)
    {
        INV_SIGMA[i * P + i] = 1.0 / ((*SIGMA)[i]);
    }
    double *U_T = (double *)malloc(sizeof(double) * P * P);
    s_transpose(*U, P, P, U_T);
    //first, multiply INV-SIGMA X U_T |=(NXP)
    double *product = (double *)malloc(sizeof(double) * N * P);
    s_multiply(INV_SIGMA, N, P, U_T, P, P, product);
    //now, multiply product X D_T |=(NXN)
    s_multiply(product, N, P, D_T, P, N, *V_T);
    free(INV_SIGMA);
    free(U_T);
    free(product);
}

double s_matrix_similarity_fabs(double *M_1, int m, int n, double *M_2)
{
    double l2_diff = 0.0;
    for (int i = 0; i < m; i++)
    {
        for (int j = 0; j < n; j++)
        {
            l2_diff += (fabs(M_1[i * n + j]) - fabs(M_2[i * n + j])) * (fabs(M_1[i * n + j]) - fabs(M_2[i * n + j]));
        }
    }
    l2_diff = sqrt(l2_diff);
    return l2_diff;
}


double s_matrix_similarity(double *M_1, int m, int n, double *M_2)
{
    double l2_diff = 0.0;
    for (int i = 0; i < m; i++)
    {
        for (int j = 0; j < n; j++)
        {
            l2_diff += (M_1[i * n + j] - M_2[i * n + j]) * (M_1[i * n + j] - M_2[i * n + j]);
        }
    }
    l2_diff = sqrt(l2_diff);
    return l2_diff;
}


double s_upper_triangular_sum(double *A, int P)
{
    double sum = 0.0;
    for (int i = 0; i < P; i++)
    {
        for (int j = i + 1; j < P; j++)
        {
            sum += A[i * P + j] * A[i * P + j];
        }
    }
    return sqrt(sum);
}

void s_set_array(double *A, int P, double *a)
{
    //copying all the A-diagonal elements:
    for (int i = 0; i < P; i++)
    {
        a[i] = A[i * P + i];
    }
    //copying upper triangular A elements:
    int index = P;
    for (int i = 0; i < P; i++)
    {
        for (int j = i + 1; j < P; j++)
        {
            a[index++] = A[i * P + j];
        }
    }
}

__device__ int device_iter;

template <int BLOCK_SIZE>
__global__ void kernel_MatMul(double *A, int rA, int cA,
                              double *B, int rB, int cB, double *C)
{
    assert(cA == rB);
    int bIDx = blockIdx.x, bIDy = blockIdx.y, tIDx = threadIdx.x, tIDy = threadIdx.y;
    int row_ = bIDy * BLOCK_SIZE + tIDy;
    int col_ = bIDx * BLOCK_SIZE + tIDx;
    __shared__ double A_sub[BLOCK_SIZE][BLOCK_SIZE];
    __shared__ double B_sub[BLOCK_SIZE][BLOCK_SIZE];
    double C_sub = 0.0;
    for (int m = 0; m < (BLOCK_SIZE + cA - 1) / BLOCK_SIZE; m++)
    {
        if (m * BLOCK_SIZE + tIDx < cA && row_ < rA)
        {
            A_sub[tIDy][tIDx] = A[row_ * cA + m * BLOCK_SIZE + tIDx];
        }
        else
        {
            A_sub[tIDy][tIDx] = 0.0;
        }
        if (m * BLOCK_SIZE + tIDy < rB && col_ < cB)
        {
            B_sub[tIDy][tIDx] = B[(m * BLOCK_SIZE + tIDy) * cB + col_];
        }
        else
        {
            B_sub[tIDy][tIDx] = 0.0;
        }
        __syncthreads();
#pragma unroll
        for (int k = 0; k < BLOCK_SIZE; k++)
            C_sub += A_sub[tIDy][k] * B_sub[k][tIDx];
        __syncthreads();
    }
    if (row_ < rA && col_ < cB)
    {
        C[cB * BLOCK_SIZE * bIDy + BLOCK_SIZE * bIDx + cB * tIDy + tIDx] = C_sub;
    }
}

__device__ void chess_tourney_params(int P, int *row_pair, int iter)
{
    //NOTE: here, row_pair is thread-local
    int localID = threadIdx.x;
    int index1, index2;
    index1 = (localID + iter) % (P - 1);
    if (localID != 0)
    {
        index2 = (P - localID + iter - 1) % (P - 1);
    }
    else
    {
        index2 = P - 1;
    }
    row_pair[0] = min(index1, index2);
    row_pair[1] = max(index1, index2);
}

__global__ void kernel_compute_all_chess_params(int P, int *device_IterBlockToElem)
{
    int blockID = blockIdx.x; 
    //each ONE of the P-1 blocks is responsible for computing chess-tourney parameters for ONE of the P-1 iterations
    int index = blockID*P + threadIdx.x*2;
    assert(threadIdx.x < P/2);
    int *row_pair = (int *) malloc(sizeof(int)*2);
    chess_tourney_params(P, row_pair, blockID);
    device_IterBlockToElem[index] = row_pair[0]; //|=(P-1)X(P/2*2)
    device_IterBlockToElem[index+1] = row_pair[1];
    free(row_pair);
}

__global__ void kernel_compute_params(double *device_A, int P, int iter, double *device_sine, double *device_cosine, int *device_IterBlockToElem)
{
    /*1 Block, P/2 threads: threadID t handles params for its alloted pair (for a particular device_iter)*/
    int localID = threadIdx.x;
    assert(localID < P / 2);
    int k, l;
    double elem, y, d, r, c, s; //,t
    k = device_IterBlockToElem[iter*P+localID*2]; //row
    l = device_IterBlockToElem[iter*P+localID*2+1]; //col
    elem = device_A[k * P + l];
    y = (device_A[l * P + l] - device_A[k * P + k]) * 0.5;
    d = fabs(y) + sqrt(elem * elem + y * y);
    r = sqrt(elem * elem + d * d);
    if (r < EPSILON)
    {
        c = 1.0;
        s = 0.0;
    }
    else
    {
        c = d / r;
        s = y / fabs(y) * elem / r; //t=y/fabs(y)*p*p/d;
    }
    device_cosine[k * P + l] = c;
    device_sine[k * P + l] = s;
}

__global__ void kernel_row_update(int iter, double *device_A, double *device_X, int P, double *device_sine, double *device_cosine, int *device_IterBlockToElem)
{
    int localID = threadIdx.x;
    int blockID = blockIdx.x;

    /*Based on blockID [total blocks=P/2], compute the corresponding two rows: p,q for device_iter*/
    __shared__ int row_pair[2];
    __shared__ double params[2]; //[sin_, cos_]
    
    if (localID == 0)            //to minimize global memory access latency at the cost of divergence
    {
        row_pair[0] = device_IterBlockToElem[iter*P+blockID * 2];
        row_pair[1] = device_IterBlockToElem[iter*P+blockID * 2 + 1];
        params[0] = device_sine[row_pair[0] * P + row_pair[1]];
        params[1] = device_cosine[row_pair[0] * P + row_pair[1]];
    }
    __syncthreads(); //all "P" threads in the block are synchronized and have access to row_pair(k,l) and params

    //CHECKPOINT: Can you reduce shared-memory bank conflicts here? 
    int k = row_pair[0], l = row_pair[1];
    double sin_ = params[0], cos_ = params[1], elem_k=device_A[k*P+localID], elem_l=device_A[l * P + localID];

    /*Concurrent modifications to all row pairs(k,l) [different blocks]*/
    /*Concurrent modifications to different-column elements of a row pair: ["P" threads of the block]*/

    /*X is col-major, i.e. write in X-transpose*/
    device_X[localID * P + k] = elem_k * cos_ - elem_l * sin_;
    device_X[localID * P + l] = elem_k * sin_ + elem_l * cos_;
}

__global__ void kernel_col_update(int iter, double *device_A, double *device_X, int P, 
    double *device_eigenvectors, double *device_sine, double *device_cosine, int *device_IterBlockToElem)
{
    int localID = threadIdx.x;
    int blockID = blockIdx.x;

    /*Based on blockID [total blocks=P/2], compute the corresponding two cols: p,q for device_iter*/
    __shared__ int col_pair[2];
    __shared__ double params[2]; //[sin_, cos_]
    if (localID == 0)            //to minimize global memory access latency at the cost of divergence
    {
        col_pair[0] = device_IterBlockToElem[iter*P+blockID * 2];
        col_pair[1] = device_IterBlockToElem[iter*P+blockID * 2 + 1];
        params[0] = device_sine[col_pair[0] * P + col_pair[1]];
        params[1] = device_cosine[col_pair[0] * P + col_pair[1]];
    }
    __syncthreads(); //all "P" threads in the block are synchronized and have access to row_pair(k,l) and params

    //CHECKPOINT: Can you reduce shared-memory bank conflicts here? Is this better than computing pair(p,q) all over again
    int k = col_pair[0], l = col_pair[1];
    double sin_ = params[0], cos_ = params[1];

    /*Concurrent modifications to all row pairs(k,l) [different blocks]*/
    /*Concurrent modifications to different-column elements of a row pair: ["P" threads of the block]*/
    double new_eigen_k, new_eigen_l;

    /* col-wise access (inefficient):*/
    //device_A[localID * P + k] = device_X[k * P + localID] * cos_ - device_X[l * P + localID] * sin_;
    //device_A[localID * P + l] = device_X[k * P + localID] * sin_ + device_X[l * P + localID] * cos_;
    //new_eigen_k = device_eigenvectors[localID * P + k]*cos_ - device_eigenvectors[localID*P+l]*sin_;
    //new_eigen_l = device_eigenvectors[localID * P+k]*sin_ + device_eigenvectors[localID*P+l]*cos_;
    //device_eigenvectors[localID * P + k] = new_eigen_k;
    //device_eigenvectors[localID * P+l] = new_eigen_l;

    /*row-wise access (efficient):*/
    int kp = k*P + localID, lp = l *P+localID;
    device_A[kp] = device_X[kp] * cos_ - device_X[lp] * sin_;
    device_A[lp] = device_X[kp] * sin_ + device_X[lp] * cos_;
    new_eigen_k = device_eigenvectors[kp]*cos_ - device_eigenvectors[lp]*sin_;
    new_eigen_l = device_eigenvectors[kp]*sin_ + device_eigenvectors[lp]*cos_;
    device_eigenvectors[kp] = new_eigen_k;
    device_eigenvectors[lp] = new_eigen_l;
}

double compute_offset(double *A, int P)
{
    double sum = 0.0;
    for (int i = 0; i < P; i++)
    {
        for (int j = i + 1; j < P; j++)
        {
            sum += fabs(A[i * P + j]);
        }
    }
    return sum;
}

double findmaxUT(double *A, int P)
{
    double temp = -1;
    for (int i = 0; i < P; i++)
    {
        for (int j = i + 1; j < P; j++)
        {
            temp = max(temp, fabs(A[i * P + j]));
        }
    }
    return temp;
}

void GPU_multiply(double *d_A, const int rA, const int cA, double *d_B, const int rB, const int cB, double *d_C, int block_size)
{
    dim3 threads(block_size, block_size);
    int gridX, gridY;
    if (cB % threads.x==0)
        gridX = cB/threads.x;
    else
        gridX = ceil(cB*1.0/threads.x);
    if (rA % threads.y==0)
        gridY = rA/threads.y;
    else
        gridY = ceil(rA*1.0/threads.y);
    
    dim3 grid(gridX, gridY);
    if (block_size == 32)
    {
        kernel_MatMul<32><<<grid, threads>>>(d_A, rA, cA, d_B, rB, cB, d_C);
    }
    else
    {
        kernel_MatMul<16><<<grid, threads>>>(d_A, rA, cA, d_B, rB, cB, d_C);
    }
    gpuErrchk(hipPeekAtLastError());
    hipDeviceSynchronize();
}

void GPU_compute_V(double **SIGMA, double *d_D_T, double **U, double **V_T, int N, int P, hipStream_t *stream1, hipStream_t *stream2)
{
    //V_T = INV-SIGMA * U_T * M
    //using pinned memory allocation for INV_SIGMA, U_T
    double *INV_SIGMA, *U_T;
    size_t double_NP = sizeof(double)*N*P, double_PP = sizeof(double)*P*P, double_NN = sizeof(double)*N*N;
    gpuErrchk(hipHostMalloc((void**)&INV_SIGMA, double_NP));
    //INV_SIGMA = (double *)calloc(N*P, sizeof(double)); //|=NXP
    memset(INV_SIGMA, 0, double_NP);
    for (int i=0; i<P; i++)
    {
        INV_SIGMA[i*P+i] = 1.0/((*SIGMA)[i]);
    }
    gpuErrchk(hipHostMalloc((void**)&U_T, double_PP));

    //first, multiply INV-SIGMA X U_T |=(NXP)
    double *d_INV_SIGMA, *d_U_T, *d_first_product;
    hipMalloc((void **)&d_INV_SIGMA, double_NP);
    hipMalloc((void **)&d_U_T, double_PP);
    hipMalloc((void **)&d_first_product, double_NP); 
    
    /***Stream-based parallelizing transfer and host code execution***/
    hipMemcpyAsync(d_INV_SIGMA, INV_SIGMA, double_NP, hipMemcpyHostToDevice, *stream1);
    s_transpose(*U, P, P, U_T);
    hipMemcpyAsync(d_U_T, U_T, double_PP, hipMemcpyHostToDevice, *stream2);
    hipDeviceSynchronize();

    GPU_multiply(d_INV_SIGMA, N, P, d_U_T, P, P, d_first_product, 32);
    hipFree(d_INV_SIGMA);
    hipFree(d_U_T);
    hipHostFree(INV_SIGMA);
    hipHostFree(U_T);

    //now, multiply product X D_T |=(NXN)
    double *d_V_T;
    hipMalloc((void **)&d_V_T, double_NN);
    GPU_multiply(d_first_product, N, P, d_D_T, P, N, d_V_T, 32);
    hipMemcpy(*V_T, d_V_T, double_NN, hipMemcpyDeviceToHost);
    hipFree(d_first_product);
    hipFree(d_V_T);
}

void SVD_and_PCA(int N,
                 int P,
                 double *D,
                 double **U,
                 double **SIGMA,
                 double **V_T,
                 int *SIGMAm,
                 int *SIGMAn,
                 double **D_HAT,
                 int *K,
                 int retention)
{
    /****************GPU-PARALLELIZED JACOBI EIGENVALUE ALGORITHM:****************/
    *SIGMAm = P;  *SIGMAn = N;
	*U = (double*) malloc(sizeof(double) * P*P);
	*SIGMA = (double*) malloc(sizeof(double) * P);
	*V_T = (double*) malloc(sizeof(double) * N*N);

    /*1.Perform SVD for D_T*/
    // Get eigen-values & eigen-vectors for D_T*D
    //hipSetDevice(5);
    high_resolution_clock::time_point t_begin, t_end, t1, t2,t3;
    t_begin = high_resolution_clock::now();
    //size_t limit = 0;
    hipStream_t stream1, stream2;
    hipStreamCreate(&stream1);
    hipStreamCreate(&stream2);

    size_t double_PP = sizeof(double)*P*P, double_NP = sizeof(double)*N*P;

    double *D_T, *A, *eigenvectors, *eigenvectors_T; //host-side (pinnable memory)
    gpuErrchk(hipHostMalloc((void**)&D_T, double_NP));
    duration<double> time_span, time_span2;
    double *device_D_T;
    gpuErrchk(hipMalloc((void **)&device_D_T, double_NP));

    int *device_IterBlockToElem; //to store mapping of P/2 "blocks" to element at (p,q), computed in the first kernel call
    gpuErrchk(hipMalloc((void **)&device_IterBlockToElem, sizeof(int) *(P-1)*P / 2 * 2));

    /********STREAM PARALLELIZATION********/
    kernel_compute_all_chess_params<<<P-1, P/2, 0, stream1>>>(P, device_IterBlockToElem);
    s_transpose(D, N, P, D_T);
    hipMemcpyAsync(device_D_T, D_T, double_NP, hipMemcpyHostToDevice, stream2);
    
    /*********Implicit Stream Barrier**********/
    hipHostFree(D_T); //D_T is not required to be stored on the host once copied to the device (~10GB)
    gpuErrchk(hipHostMalloc((void**)&A, double_PP));
    eigenvectors = (double *)malloc(double_PP);
    gpuErrchk(hipHostMalloc((void**)&eigenvectors_T, double_PP));
    double *device_D, *device_A, *device_X;
    gpuErrchk(hipMalloc((void **)&device_D, double_NP));
    gpuErrchk(hipMalloc((void **)&device_A, double_PP));
    gpuErrchk(hipMalloc((void **)&device_X, double_PP));
    gpuErrchk(hipMemcpy(device_D, D, double_NP, hipMemcpyHostToDevice));

    //printf("starting multiplication of D_T*D=A:\n");

    /* Parallelized matrix multiplication (D_T*D=A) */
    GPU_multiply(device_D_T, P, N, device_D, N, P, device_A, 32);
    
    s_initialize_identity(eigenvectors_T, P);
    double *device_eigenvectors_T;
    gpuErrchk(hipMalloc((void **)&device_eigenvectors_T, double_PP));
    hipMemcpy(device_eigenvectors_T, eigenvectors_T, double_PP, hipMemcpyHostToDevice);

    double *device_sine, *device_cosine;
    gpuErrchk(hipMalloc((void **)&device_sine, double_PP));
    gpuErrchk(hipMalloc((void **)&device_cosine, double_PP));
    hipMemset(device_sine, 0, double_PP);
    hipMemset(device_cosine, 0, double_PP);
   
    int grid_size, block_size=P, iter = 0, counter = 0;
    double offset_ = THRESHOLD + 1;
    if (P%2==0)
        grid_size = P / 2;
    else
        grid_size = P/2+1;
   
    while (counter < MAX_SWEEPS && offset_ > THRESHOLD) //sweeps
    {
        iter = 0;
        while (iter < P - 1)
        {
            //Compute rotation parameters for all (p,q): q>p
            kernel_compute_params<<<1, grid_size>>>(device_A, P, iter, device_sine, device_cosine, device_IterBlockToElem);
            hipDeviceSynchronize();
            
            //row-update kernel
            kernel_row_update<<<grid_size, block_size>>>(iter, device_A, device_X, P, device_sine, device_cosine, device_IterBlockToElem);
            hipDeviceSynchronize();

            //col-update & eigen-vector update kernel
            kernel_col_update<<<grid_size, block_size>>>(iter, device_A, device_X, P, device_eigenvectors_T, device_sine, device_cosine, device_IterBlockToElem);
            hipDeviceSynchronize();
            iter++;
        }
        hipMemcpy(A, device_A, double_PP, hipMemcpyDeviceToHost);
        
        offset_ = compute_offset(A, P);
        printf("Sweep:%d, offset:%f\n", counter, offset_);
        counter++;
    }
    
    hipMemcpy(eigenvectors_T, device_eigenvectors_T, double_PP, hipMemcpyDeviceToHost);
    s_transpose(eigenvectors_T, P, P, eigenvectors);
    
    double *eigenvalues = (double *)malloc(sizeof(double) * P);
    int *e_indices = (int *)malloc(sizeof(int) * P);
    for (int i = 0; i < P; i++)
    {
        eigenvalues[i] = A[i * P + i];
        e_indices[i] = i;
	}

    //sort eigenvalues in desc:
    s_mergesort(eigenvalues, P, e_indices, 0, P - 1);
    double temp_ = eigenvalues[0];
    for (int i = 0; i < P; i++)
    {
        // printf("%f,", eigenvalues[i]);
        assert(temp_>=eigenvalues[i]);
        temp_=eigenvalues[i];
    }
   
    hipError_t err = hipGetLastError();
    if (err != hipSuccess)
    {
        printf("Error: %s\n", hipGetErrorString(err));
    }
    
    //computing SIGMA:
    double sum_variance = 0.0;
    for (int i = 0; i < P; i++)
    {
        sum_variance+=eigenvalues[i];
        (*SIGMA)[i] = sqrt(eigenvalues[i]);
    }
	
    //computing U:
    int index;
    for (int row = 0; row < P; row++)
    {
        index = row*P;
        for (int col = 0; col < P; col++)
        {
            (*U)[index + col] = eigenvectors[index + e_indices[col]];
        }
    }

    //compute V_T:
    GPU_compute_V(SIGMA, device_D_T, U, V_T, N, P, &stream1, &stream2);
    
    //Parallelized (PCA):
    int K_ = 0;
    double retention_ = 0.0;
    int count_ = 0;
    while((retention_<retention) && (count_ < P))
    {
        retention_+=(eigenvalues[count_]/sum_variance)*100;
        K_++;
        count_++;
    }
    *K = K_;
    size_t double_PK = sizeof(double)*P*K_, double_NK = sizeof(double)*N*K_;
    printf("K GPU:%d, retention:%f\n", *K, retention_);
    double *W; //= (double *)malloc(sizeof(double)*P*K_);
    gpuErrchk(hipHostMalloc((void**)&W, double_PK));
    *D_HAT = (double *)malloc(double_NK);
    int index_, index_2;
    for (int r=0; r<P; r++)
    {
        index_ = r*K_; index_2 = r*P;
        for (int c=0; c<K_; c++)
        {
            W[index_+c] = (*U)[index_2+c];
        }
    }

    //now, multiply D*W |=(NxP.PxK=NxK)
    double *device_W, *device_D_HAT;
    gpuErrchk(hipMalloc((void **)&device_W, double_PK));
    gpuErrchk(hipMalloc((void**)&device_D_HAT, double_NK));
    gpuErrchk(hipMemcpy(device_W, W, double_PK, hipMemcpyHostToDevice));
    hipHostFree(W);
    GPU_multiply(device_D, N, P, device_W, P, K_, device_D_HAT, 32);
    hipMemcpy(*D_HAT, device_D_HAT, double_NK, hipMemcpyDeviceToHost);
    
    t_end = high_resolution_clock::now();
    time_span = duration_cast<duration<double>>(t_end - t_begin);
    printf("TOTAL TIME:%f\n", time_span.count());    
    return;

    /****************SERIAL JACOBI EIGENVALUE ALGORITHM (can be used for Speedup Computation):****************/
    // t_begin = high_resolution_clock::now();
    // //begin Jacobi eigenvalue algorithm:
    // int state = P, num_iter = 0, m, k, l; //m: pivot row identifier
    // double p, y, d, r, c, s, t;           //p: pivot element, c: cos, s: sin
    // double *E = (double *)malloc(sizeof(double)*P*P);
    // s_initialize_identity(E, P); //P*P
    // double *E_ = (double *)malloc(sizeof(double) * P * P);
    // double *e = (double *)malloc(sizeof(double) * P); //init eigen-values array
    // double *e_ = (double *)malloc(sizeof(double) * P);
    // int *ind = (int *)malloc(sizeof(int) * P);        //init maxindex array
    // bool *changed = (bool *)malloc(sizeof(bool) * P); //change in eigen_value[k]
    // double *A_s = (double *)calloc(P * P, sizeof(double));
    // D_T = (double *)malloc(sizeof(double) * P * N);
    // s_transpose(D, N, P, D_T);
    // s_multiply(D_T, P, N, D, N, P, A_s);

    // printf("printing A_s:\n");
    // //print_matrix(A_s, P, P, 1);
    // for (int i = 0; i < P; i++)
    // {
    //     ind[i] = s_maxind(A_s, P, i); //NOTE: undefined for last row
    //     e[i] = A_s[i * P + i];
    //     changed[i] = true;
    //     //printf("%d, %d\n", i, ind[i]);
    // }
    // while (state && num_iter < MAX_ITER)
    // {
    //     memcpy(E_, E, sizeof(double) * P * P);
    //     memcpy(e_, e, sizeof(double) * P);
    //     //find index (k,l) of pivot p
    //     m = 0;
    //     for (int i = 1; i < P - 1; i++)
    //     {
    //         //printf("i:%d, %d, %f\n", i, ind[i], A[i*P+ind[i]]);
    //         if (fabs(A_s[i * P + ind[i]]) > fabs(A_s[m * P + ind[m]]))
    //         {
    //             m = i;
    //         }
    //     }
    //     k = m;
    //     l = ind[k];
    //     p = A_s[k * P + l];
    //     y = 0.5 * (e[l] - e[k]);
    //     d = fabs(y) + sqrt(p * p + y * y);
    //     r = sqrt(p * p + d * d);
    //     c = d / r;
    //     s = p / r;
    //     t = p * p / d;
    //     if (y < 0)
    //     {
    //         s = -s;
    //         t = -t;
    //     }
    //     A_s[k * P + l] = 0.0;
    //     s_update(k, -t, e, changed, &state);
    //     s_update(l, t, e, changed, &state);

    //     //rotate rows and cols k and l:
    //     for (int i = 0; i < k; i++)
    //     {
    //         s_rotate(i, k, i, l, A_s, P, c, s);
    //     }
    //     for (int i = k + 1; i < l; i++)
    //     {
    //         s_rotate(k, i, i, l, A_s, P, c, s);
    //     }
    //     for (int i = l + 1; i < P; i++)
    //     {
    //         s_rotate(k, i, l, i, A_s, P, c, s);
    //     }
    //     //rotate eigenvectors:
    //     for (int i = 0; i < P; i++)
    //     {
    //         double e_ik = c * E[i * P + k] - s * E[i * P + l];
    //         double e_il = s * E[i * P + k] + c * E[i * P + l];
    //         E[i * P + k] = e_ik;
    //         E[i * P + l] = e_il;
    //     }
    //     ind[k] = s_maxind(A_s, P, k);
    //     ind[l] = s_maxind(A_s, P, l);
    //     double diff = l2_diff_norm(e_, e, P);
    //     double diff_2 = l2_matrix_diff_norm(E_, E, P, P);
    //     double upper_triangular_sum = s_upper_triangular_sum(A_s, P);
    //     printf("\rITER:%d, state:%d, diff:%.10f up-sum:%f", num_iter, state, diff + diff_2, upper_triangular_sum);
    //     fflush(stdout);
    //     num_iter++;
    // }
    // //sort eigenvalues in desc:
    // int *indices = (int *)malloc(sizeof(int) * P);
    // for (int i = 0; i < P; i++)
    // {
    //     indices[i] = i;
    // }
    // s_mergesort(e, P, indices, 0, P - 1);
    // printf("Indices arr:\n");
    // for (int i = 0; i < P; i++)
    // {
    //     printf("%d,", indices[i]);
    // }
    // printf("\n");
    // printf("e arr:\n");
    // for (int i = 0; i < P; i++)
    // {
    //     printf("%f,", e[i]);
    // }
    // printf("\n");

    // // //computing SIGMA:
    // // printf("printing sigma:\n");
    // double sum_eigenvalues_s=0.0;
    // for (int i = 0; i < P; i++)
    // {
    //     (*SIGMA)[i] = sqrt(e[i]);
    //     sum_eigenvalues_s+=e[i];
    //     //printf("%f,", (*SIGMA)[i]);
    // }
    // printf("sum evals_s:%f\n", sum_eigenvalues_s);
    // printf("\n");
    // //computing SIGMA_MATRIX:
    // double *temp_sigma = (double *)calloc(P * N, sizeof(double));
    // for (int i = 0; i < P; i++)
    // {
    //     //assert(e[i]>=0);
    //     temp_sigma[i * N + i] = sqrt(e[i]);
    // }

    // //eigenvectors matrix (U for D_T*D):
    // printf("printing E:\n");

    // //L2
    // double sum_temp=0.0;
    // for (int x=0; x<P; x++)
    // {
    //     for (int y=0; y<P; y++)
    //     {
    //         sum_temp+=(fabs(E[x*P+indices[y]])-fabs(eigenvectors[x*P+e_indices[y]]))*(fabs(E[x*P+indices[y]])-fabs(eigenvectors[x*P+e_indices[y]]));
    //     }
    // }
    // printf("L-2 fabs diff in E:%f\n", sqrt(sum_temp));
    
    // printf("printing U:\n");
    // double *u_s = (double *) malloc(sizeof(double)*P*P);
    // for (int row = 0; row < P; row++)
    // {
    //     for (int col = 0; col < P; col++)
    //     {
    //         // (*U)[row * P + col] = E[row * P + indices[col]];
    //         u_s[row * P + col] = E[row * P + indices[col]];
    //        // printf("%f,", (*U)[row*P+col]);
    //     }
    //     //printf("\n");
    // }
    // //compute V_T:
    // double *V_T_s = (double *)calloc(N*N, sizeof(double));
    // s_compute_V(SIGMA, D_T, &u_s, &V_T_s, N, P);
   
    // printf("\nprinting V_T:\n");
    // double sim1=s_matrix_similarity_fabs(*U, P, P, u_s);
    // printf("L2-matrix fabs sim bw U's:%.10f\n", sim1);
    // double sim2 = s_matrix_similarity_fabs(*V_T, N, N, V_T_s);
    // printf("L2-matrix fabs sim bw V_T's:%.10f\n", sim2);
    // sim2 = s_matrix_similarity_fabs(*V_T, N, N, *V_T);
    // printf("L2-matrix fabs sim bw V_Tg's same:%.10f\n", sim2);
    // printf("prinitng V_t_s:\n");
    // //print_matrix(V_T_s, N, N, 1);
    
    // //compute serial PCA:
    //  int K_s=0;
    //  double retention_s = 0.0;
    //  int count_s = 0;
    //  while((retention_s<retention) && (count_s < P))
    //  {
    //      retention_s+=((*SIGMA)[count_s]*(*SIGMA)[count_s]/sum_eigenvalues_s)*100;
    //      K_s++;
    //      count_s++;
    //  }
    //  printf("K_s CPU:%d, retention_S:%f\n", K_s, retention_s);
    //  assert(*K==K_s);
    //  double *W_s = (double *)malloc(sizeof(double)*P*K_s);
    //  double *D_HAT_s = (double *)malloc(sizeof(double)*N*K_s);
    //  for (int r=0; r<P; r++)
    //  {
    //      for (int c=0; c<K_s; c++)
    //      {
    //          W_s[r*K_s+c] = u_s[r*P+c];
    //      }
    //  }
 
    //  //now, serially multiply D*W |=(NxP.PxK=NxK)
    // s_multiply(D, N, P, W_s, P, K_s, D_HAT_s);
    // sim2 = s_matrix_similarity_fabs(D_HAT_s, N, K_s, *D_HAT);
    // printf("L2-matrix fabs sim bw PCAs:%.10f\n", sim2);
    // sim2 = s_matrix_similarity_fabs(*D_HAT, N, K_s, *D_HAT);
    // printf("L2-matrix fabs sim bw same G PCAs:%.10f\n", sim2);
    // t_end = high_resolution_clock::now();
    // time_span = duration_cast<duration<double>>(t_end - t_begin);
    // printf("SEQUENTIAL TOTAL TIME:%f\n print matrix", time_span.count());    
    // print_matrix(D_T, P, N, 1);
    // return;   
}
